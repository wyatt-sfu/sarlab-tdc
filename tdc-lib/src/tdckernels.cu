#include "hip/hip_runtime.h"
/* Standard library headers */
#include <algorithm>
#include <cstddef>

/* CUDA headers */
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_reduce.cuh>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/driver_types.h>
#include <hip/hip_math_constants.h>
#include <hip/hip_vector_types.h>

/* Project headers */
#include "gpumath.h"
#include "tuning.h"

/* Class header */
#include "tdckernels.h"

#define SPEED_OF_LIGHT_F 299792458.0F

/* Global variable used for storing the maximum of the window array */
__device__ float2 SumResults[NUM_STREAMS];

/**
 * Cuda kernel for initializing the range window array.
 *
 * rgWin: 1D array that will be filled with the range window weights
 * nSamples: Number of samples
 */
__global__ void initRangeWindowKernel(float *__restrict__ rgWin, int nSamples)
{
    unsigned int const sampleIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (sampleIdx >= nSamples) {
        return;
    }

    rgWin[sampleIdx] = RANGE_WINDOW_A_PARAMETER
                       - ((1.0 - RANGE_WINDOW_A_PARAMETER)
                          * cosf(2.0F * HIP_PI_F * sampleIdx / nSamples));
}

/**
 * Initialize the range window array on the GPU
 *
 * rgWin: 1D array that will be filled with the range window weights
 * nSamples: Number of samples
 */
void initRangeWindow(float *__restrict__ rgWin, int nSamples)
{
    dim3 const blockSize(32, 1, 1);
    dim3 const gridSize((nSamples + blockSize.x - 1) / blockSize.x, 1, 1);
    initRangeWindowKernel<<<gridSize, blockSize>>>(rgWin, nSamples);
}

__global__ void dopplerCentroid(float4 const *velocity, float4 const *attitude,
                                float lambda, int chunkIdx, int nPri, int nSamp)
{
    float lambdaFac = 2.0f / lambda;
}

/**
 * Cuda kernel for computing the window function for the specified chunk of
 * raw data.
 */
__global__ void createWindowKernel(
    // Window arrays
    float *__restrict__ window, // 2D full window to apply to raw data chunk
    float const *__restrict__ rangeWindow, // 1D range window

    // Positioning data
    float3 const *__restrict__ velocity, // [m] 2D, x,y,z velocity at each PRI/sample
    float4 const *__restrict__ attitude, // 2D quaternion at each PRI/sample

    // Radar parameters
    float lambda, // [m] Radar carrier wavelength
    float dopplerBw, // [Hz] Bandwidth for windowing

    // Data shape arguments
    int chunkIdx, // Current chunk index
    int nPri, // Number of PRIs in the full acquisition
    int nSamples // Number of samples per PRI
)
{
    unsigned int const priChunkIdx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int const sampleIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int const priGlobalIdx = chunkIdx * PRI_CHUNKSIZE + priChunkIdx;

    if (priGlobalIdx < nPri && priChunkIdx < PRI_CHUNKSIZE && sampleIdx < nSamples) {
        float lambdaFac = 2.0f / lambda;
        // Compute the Doppler centroid for this pulse

        // TODO: Compute window based on Doppler
        window[priChunkIdx * nSamples + sampleIdx] = rangeWindow[sampleIdx];
    }
}

/**
 * Wrapper around the cuda kernel createWindowKernel
 */
void createWindow(
    // Window arrays
    float *__restrict__ window, // 2D full window to apply to raw data chunk
    float const *__restrict__ rangeWindow, // 1D range window

    // Positioning data
    float3 const *__restrict__ velocity, // [m] 2D, x,y,z velocity at each PRI/sample
    float4 const *__restrict__ attitude, // 2D quaternion at each PRI/sample

    // Radar parameters
    float lambda, // [m] Radar carrier wavelength
    float dopplerBw, // [Hz] Bandwidth for windowing

    // Data shape arguments
    int chunkIdx, // Current chunk index
    int nPri, // Number of PRIs in the full acquisition
    int nSamples, // Number of samples per PRI
    hipStream_t stream // Stream to run the kernel in
)
{
    dim3 const blockSize(WindowKernel::BlockSizeX, WindowKernel::BlockSizeY, 1);
    dim3 const gridSize((nSamples + blockSize.x - 1) / blockSize.x,
                        (PRI_CHUNKSIZE + blockSize.y - 1) / blockSize.y, 1);
    createWindowKernel<<<gridSize, blockSize, 0, stream>>>(
        window, rangeWindow, velocity, attitude, lambda, dopplerBw, chunkIdx, nPri,
        nSamples);
}

/**
 * Cuda kernel which computes the reference response from a target at the
 * specified position.
 */
__global__ void referenceResponseKernel(
    // Data array parameters
    float2 *__restrict__ reference, // 2D, IQ data to contain reference data
    float const *__restrict__ window, // 2D full window to apply to raw data chunk
    float3 const *__restrict__ position, // [m] 2D, x,y,z position at each PRI/sample
    float const *__restrict__ sampleTimes, // [s] 1D, Time of each sample in a PRI
    float3 target, // [m] Location on the focus grid

    // Radar operating parameters
    float startFreq, // [Hz] PRI start frequency
    float modRate, // [Hz/s] Modulation rate

    // Data shape arguments
    int chunkIdx, // Current chunk index
    int nPri, // Number of PRIs in the full acquisition
    int nSamples // Number of samples per PRI
)
{
    unsigned int const priChunkIdx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int const sampleIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int const priGlobalIdx = chunkIdx * PRI_CHUNKSIZE + priChunkIdx;
    const ptrdiff_t elementIdx =
        static_cast<ptrdiff_t>(priChunkIdx) * nSamples + sampleIdx;

    if (priGlobalIdx < nPri && priChunkIdx < PRI_CHUNKSIZE && sampleIdx < nSamples) {

        const float3 phase_centre = position[elementIdx];
        const float winVal = window[elementIdx];
        float dist_to_target =
            norm3df(phase_centre.x - target.x, phase_centre.y - target.y,
                    phase_centre.z - target.z);

        const float freq = fmaf(sampleTimes[sampleIdx], modRate, startFreq);
        const float phi = -4.0F * dist_to_target * (freq / SPEED_OF_LIGHT_F);
        float sinval = 0.0;
        float cosval = 0.0;
        sincospif(phi, &sinval, &cosval);
        reference[elementIdx] = {cosval * winVal, sinval * winVal};
    }
}

void referenceResponse(
    // Data array parameters
    float2 *__restrict__ reference, // 2D, IQ data to contain reference data
    float const *__restrict__ window, // 2D full window to apply to raw data chunk
    float3 const *__restrict__ position, // [m] 2D, x,y,z position at each PRI/sample
    float const *__restrict__ sampleTimes, // [s] 1D, Time of each sample in a PRI
    float3 target, // [m] Location on the focus grid

    // Radar operating parameters
    float startFreq, // [Hz] PRI start frequency
    float modRate, // [Hz/s] Modulation rate

    // Data shape arguments
    int chunkIdx, // Current chunk index
    int nPri, // Number of PRIs in the full acquisition
    int nSamples, // Number of samples per PRI
    hipStream_t stream // Stream to run the kernel in
)
{
    dim3 const refBlockSize(ReferenceResponseKernel::BlockSizeX,
                            ReferenceResponseKernel::BlockSizeY, 1);
    dim3 const refGridSize((nSamples + refBlockSize.x - 1) / refBlockSize.x,
                           (PRI_CHUNKSIZE + refBlockSize.y - 1) / refBlockSize.y, 1);

    referenceResponseKernel<<<refGridSize, refBlockSize, 0, stream>>>(
        reference, window, position, sampleTimes, target, startFreq, modRate, chunkIdx,
        nPri, nSamples);
}

/**
 * Correlate the raw data with the reference response.
 */
__global__ void correlateWithReference(
    // Argument list
    const float2 *__restrict__ raw, // Raw radar data
    float2 *__restrict__ reference, // Reference response. Correlation is
                                    // written back into this array.
    int chunkIdx, // Current chunk index
    int nPri, // Number of PRIs
    int nSamples // Number of samples in a PRI
)
{
    unsigned int const priChunkIdx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int const sampleIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int const priGlobalIdx = chunkIdx * PRI_CHUNKSIZE + priChunkIdx;
    const ptrdiff_t elementIdx =
        static_cast<ptrdiff_t>(priChunkIdx) * nSamples + sampleIdx;

    if (priGlobalIdx < nPri && priChunkIdx < PRI_CHUNKSIZE && sampleIdx < nSamples) {
        const float2 v1 = raw[elementIdx];
        float2 v2 = reference[elementIdx];

        v2.y *= -1.0; // conjugate
        reference[elementIdx].x = (v1.x * v2.x) - (v1.y * v2.y);
        reference[elementIdx].y = (v1.x * v2.y) + (v1.y * v2.x);
    }
}

__global__ void addToImage(float2 *__restrict__ pixel, float2 *__restrict__ sumVal)
{
    *pixel = (*pixel) + (*sumVal);
}

/**
 * Correlate the raw data with the reference array and put the result in the
 * focused image
 */
void correlateAndSum(
    // Data array parameters
    float2 const *__restrict__ raw, // 2D, IQ data chunk
    float2 *__restrict__ reference, // 2D, Reference response to correlate with
    void *__restrict__ scratch, // Scratch space for sum reduction
    size_t scratchSize, // Size of sum scratch space

    // Focus image
    float2 *__restrict__ pixel, // Pointer to the current pixel

    // Data shape arguments
    int chunkIdx, // Current chunk index
    int nPri, // Number of PRIs in the full acquisition
    int nSamples, // Number of samples per PRI
    int streamIdx, // Stream index
    hipStream_t stream // Stream to run the kernel in
)
{
    // First correlate the reference and raw data
    dim3 const blockSize(CorrelateKernel::BlockSizeX, CorrelateKernel::BlockSizeY, 1);
    dim3 const gridSize((nSamples + blockSize.x - 1) / blockSize.x,
                        (PRI_CHUNKSIZE + blockSize.y - 1) / blockSize.y, 1);
    correlateWithReference<<<gridSize, blockSize, 0, stream>>>(raw, reference, chunkIdx,
                                                               nPri, nSamples);

    // Then sum the result
    void *devPtr;
    hipGetSymbolAddress(&devPtr, HIP_SYMBOL(SumResults));
    float2 *sumResult = reinterpret_cast<float2 *>(devPtr) + streamIdx;
    size_t priIndex = chunkIdx * PRI_CHUNKSIZE;
    size_t prisToSum = std::min(PRI_CHUNKSIZE, nPri - priIndex);
    hipcub::DeviceReduce::Sum(scratch, scratchSize, reference, sumResult,
                           prisToSum * nSamples, stream);
    addToImage<<<1, 1, 0, stream>>>(pixel, sumResult);
}

/**
 * Returns the scratch size needed in bytes for the correlateAndSum function
 */
size_t sumScratchSize(int nSamples)
{
    void *scratch = nullptr;
    size_t scratchSize = 0;
    float2 *dataIn = nullptr;
    float2 dataOut = {0, 0};
    hipcub::DeviceReduce::Sum(scratch, scratchSize, dataIn, &dataOut,
                           PRI_CHUNKSIZE * nSamples);
    return scratchSize;
}
